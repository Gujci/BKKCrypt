#include "hip/hip_runtime.h"
#include "BKKCrypt.h"
#include <string.h>

#include ""
#include "hip/hip_runtime.h"


//Here is the magic, the core of the algorithm
__global__ void BKKCryptKernel(const char *d_in, char *d_out, size_t length)
{
	//every thread process one element, the index of this element:
	size_t index = threadIdx.x + blockIdx.x * blockDim.x;

	if (index < length) {
		d_out[index] = d_in[index];
	}
}


bool BKKCrypt(const char *in, char *out)
{
	char *d_in = NULL;					//pointers to the device memory
	char *d_out = NULL;

	size_t length = strlen(in) + 1;		//size of the input and (hopefully) the output array

	bool returnedValue = true;			

	//allocating device memory
	if (hipMalloc((void**)&d_in, length) != hipSuccess) {
		returnedValue = false;
		goto Error;
	}

	if (hipMalloc((void**)&d_out, length) != hipSuccess) {
		returnedValue = false;
		goto Error;
	}

	//copy the input array to the device
	if (hipMemcpy(d_in, in, length, hipMemcpyHostToDevice) != hipSuccess) {
		returnedValue = false;
		goto Error;
	}

	//computing the grid and block size.
	//we choose the block size to be equal to the numbers of threads in a warp
	//the grid size is calculated so that we can process every input elements
	hipDeviceProp_t device;
	hipGetDeviceProperties(&device, 0);
	int threadsPerBlock = device.warpSize;
	int blocks = (length + threadsPerBlock - 1) / threadsPerBlock;

	//The magic
	BKKCryptKernel <<<blocks, threadsPerBlock >>> (d_in, d_out, length);

	hipDeviceSynchronize();
	hipError_t cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		returnedValue = false;
		goto Error;
	}

	//copy the output string to the host
	if (hipMemcpy(out, d_out, length, hipMemcpyDeviceToHost) != hipSuccess) {
		returnedValue = false;
		goto Error;
	}

Error:
	//We have to clean up. Very important!!!
	hipFree(d_in);
	hipFree(d_out);
	return returnedValue;
}


